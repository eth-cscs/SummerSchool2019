#include "hip/hip_runtime.h"
// linear algebra subroutines
// Ben Cumming @ CSCS

#include <iostream>

#include <cmath>
#include <cstdio>

#include <mpi.h>

#include "linalg.h"
#include "operators.h"
#include "stats.h"
#include "data.h"

namespace linalg {

namespace kernels {
    __global__
    void fill(
            double *y,
            const double value,
            int n)
    {
        auto i = threadIdx.x + blockDim.x*blockIdx.x;

        if(i < n) {
            y[i] = value;
        }
    }

    __global__
    void axpy(
            double* y,
            const double alpha,
            const double* x,
            int n)
    {
        auto i = threadIdx.x + blockDim.x*blockIdx.x;

        if(i < n) {
            y[i] += alpha * x[i];
        }
    }

    __global__
    void add_scaled_diff(
            double *y,
            const double* x,
            const double alpha,
            const double *l,
            const double *r,
            const int n)
    {
        auto i = threadIdx.x + blockDim.x*blockIdx.x;

        if(i < n) {
            y[i] = x[i] + alpha * (l[i] - r[i]);
        }
    }

    __global__
    void scaled_diff(
            double *y,
            const double alpha,
            const double* l,
            const double* r,
            int n)
    {
        auto i = threadIdx.x + blockDim.x*blockIdx.x;

        if(i < n) {
            y[i] = alpha * (l[i] - r[i]);
        }
    }

    __global__
    void scale(
            double *y,
            const double alpha,
            const double *x,
            int n)
    {
        auto i = threadIdx.x + blockDim.x*blockIdx.x;

        if(i < n) {
            y[i] = alpha * x[i];
        }
    }

    __global__
    void lcomb(
            double *y,
            const double alpha,
            const double *x,
            const double beta,
            const double *z,
            int n)
    {
        auto i = threadIdx.x + blockDim.x*blockIdx.x;

        if(i < n) {
            y[i] = alpha * x[i] + beta * z[i];
        }
    }

    __global__
    void copy(
            double *y,
            const double* x,
            int n)
    {
        auto i = threadIdx.x + blockDim.x*blockIdx.x;

        if(i < n) {
            y[i] = x[i];
        }
    }
} // namespace kernels

bool cg_initialized = false;
Field r;
Field Ap;
Field p;
Field Fx;
Field Fxold;
Field v;
Field xold;

// block dimensions for blas 1 calls
const int block_dim = 192;
int calculate_grid_dim(const int block_dim, int n) {
    return (n+block_dim-1)/block_dim;
}

using namespace operators;
using namespace stats;
using data::Field;

// initialize temporary storage fields used by the cg solver
// I do this here so that the fields are persistent between calls
// to the CG solver. This is useful if we want to avoid malloc/free calls
// on the device for the OpenACC implementation
void cg_init(int nx, int ny)
{
    Ap.init(nx,ny);
    r.init(nx,ny);
    p.init(nx,ny);
    Fx.init(nx,ny);
    Fxold.init(nx,ny);
    v.init(nx,ny);
    xold.init(nx,ny);

    cg_initialized = true;
}

////////////////////////////////////////////////////////////////////////////////
//  blas level 1 reductions
////////////////////////////////////////////////////////////////////////////////

// computes the inner product of x and y
// x and y are vectors on length N
double ss_dot(Field const& x, Field const& y)
{
    double result = 0;
    double result_global = 0;
    const int N = x.length();

    auto status =
        hipblasDdot(
            cublas_handle(),  N,
            x.device_data(), 1,
            y.device_data(), 1,
            &result
        );

    MPI_Allreduce(&result, &result_global, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);

    return result_global;
}

// computes the 2-norm of x
// x is a vector on length N
double ss_norm2(Field const& x)
{
    double result = 0;
    double result_global = 0;
    const int n = x.length();

    auto status =
        hipblasDnrm2(
            cublas_handle(), n,
            x.device_data(), 1,
            &result
        );

    // take the square of the result, because we still have to sum of the local
    // partial sums before taking sqrt of the full global sum
    result *= result;

    MPI_Allreduce(&result, &result_global, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);

    return sqrt(result_global);
}

// sets entries in a vector to value
// x is a vector on length N
// value is th
void ss_fill(Field& x, const double value)
{
    const int n = x.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::fill<<<grid_dim, block_dim>>>(x.device_data(), value, n);
}

////////////////////////////////////////////////////////////////////////////////
//  blas level 1 vector-vector operations
////////////////////////////////////////////////////////////////////////////////

// computes y := alpha*x + y
// x and y are vectors on length N
// alpha is a scalar
void ss_axpy(Field& y, const double alpha, Field const& x)
{
    const int n = y.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::axpy<<<grid_dim, block_dim>>>
        (y.device_data(), alpha, x.device_data(), x.length());
}

// computes y = x + alpha*(l-r)
// y, x, l and r are vectors of length N
// alpha is a scalar
void ss_add_scaled_diff(Field& y, Field const& x, const double alpha,
    Field const& l, Field const& r)
{
    const int n = y.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::add_scaled_diff<<<grid_dim, block_dim>>>
        (y.device_data(), x.device_data(), alpha, l.device_data(), r.device_data(), n);
}

// computes y = alpha*(l-r)
// y, l and r are vectors of length N
// alpha is a scalar
void ss_scaled_diff(Field& y, const double alpha,
    Field const& l, Field const& r)
{
    const int n = y.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::scaled_diff<<<grid_dim, block_dim>>>
        (y.device_data(), alpha, l.device_data(), r.device_data(), n);
}

// computes y := alpha*x
// alpha is scalar
// y and x are vectors on length n
void ss_scale(Field& y, const double alpha, Field& x)
{
    const int n = x.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::scale<<<grid_dim, block_dim>>>
        (y.device_data(), alpha, x.device_data(), n);
}

// computes linear combination of two vectors y := alpha*x + beta*z
// alpha and beta are scalar
// y, x and z are vectors on length n
void ss_lcomb(Field& y, const double alpha, Field& x, const double beta,
    Field const& z)
{
    const int n = x.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::lcomb<<<grid_dim, block_dim>>>
        (y.device_data(), alpha, x.device_data(), beta, z.device_data(), n);
}

// copy one vector into another y := x
// x and y are vectors of length N
void ss_copy(Field& y, Field const& x)
{
    const int n = x.length();
    auto grid_dim = calculate_grid_dim(block_dim, n);

    kernels::copy<<<grid_dim, block_dim>>>
        (y.device_data(), x.device_data(), n);
}

// conjugate gradient solver
// solve the linear system A*x = b for x
// the matrix A is implicit in the objective function for the diffusion equation
// the value in x constitute the "first guess" at the solution
// x(N)
// ON ENTRY contains the initial guess for the solution
// ON EXIT  contains the solution
void ss_cg(Field& x, Field const& b, const int maxiters, const double tol, bool& success)
{

    // this is the dimension of the linear system that we are to solve
    int nx = data::domain.nx;
    int ny = data::domain.ny;

    if(!cg_initialized) {
        cg_init(nx,ny);
    }

    // epsilon value use for matrix-vector approximation
    double eps     = 1.e-8;
    double eps_inv = 1. / eps;

    // allocate memory for temporary storage
    ss_fill(Fx,    0.0);
    ss_fill(Fxold, 0.0);
    ss_copy(xold, x);

    // matrix vector multiplication is approximated with
    // A*v = 1/epsilon * ( F(x+epsilon*v) - F(x) )
    //     = 1/epsilon * ( F(x+epsilon*v) - Fxold )
    // we compute Fxold at startup
    // we have to keep x so that we can compute the F(x+exps*v)
    diffusion(x, Fxold);

    // v = x + epsilon*x
    ss_scale(v, 1.0 + eps, x);

    // Fx = F(v)
    diffusion(v, Fx);

    // r = b - A*x
    // where A*x = (Fx-Fxold)/eps
    ss_add_scaled_diff(r, b, -eps_inv, Fx, Fxold);

    // p = r
    ss_copy(p, r);

    // rold = <r,r>
    double rold = ss_dot(r, r);
    double rnew = rold;

    // check for convergence
    success = sqrt(rold) < tol;
    if (success) {
        return;
    }

    int iter;
    for(iter=0; iter<maxiters; iter++) {
        // Ap = A*p
        ss_lcomb(v, 1.0, xold, eps, p);
        diffusion(v, Fx);
        ss_scaled_diff(Ap, eps_inv, Fx, Fxold);

        // alpha = rold / p'*Ap
        double alpha = rold / ss_dot(p, Ap);

        // x += alpha*p
        ss_axpy(x, alpha, p);

        // r -= alpha*Ap
        ss_axpy(r, -alpha, Ap);

        // find new norm
        rnew = ss_dot(r, r);

        // test for convergence
        if (sqrt(rnew) < tol) {
            success = true;
            break;
        }

        // p = r + (rnew/rold) * p
        ss_lcomb(p, 1.0, r, rnew / rold, p);

        rold = rnew;
    }
    stats::iters_cg += iter + 1;

    if (!success && !data::domain.rank) {
        std::cerr << "ERROR: CG failed to converge after " << iter
                  << " iterations, with residual " << sqrt(rnew)
                  << std::endl;
    }
}

} // namespace linalg
