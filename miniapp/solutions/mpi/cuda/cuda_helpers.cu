
#include <hip/hip_runtime.h>
#include <iostream>

#include <cstdlib>
#include <hipblas.h>

void cuda_check_last_kernel(std::string const& errstr) {
    auto status = hipGetLastError();
    if(status != hipSuccess) {
        std::cout << "error: CUDA kernel launch :" << errstr << " : "
                  << hipGetErrorString(status) << std::endl;
        exit(-1);
    }
}

void cuda_api_call(hipError_t error_code) {
    if(error_code != hipSuccess) {
        std::cerr << "error: CUDA API call : "
                  << hipGetErrorString(error_code) << std::endl;
        exit(-1);
    }
}

hipblasHandle_t& cublas_handle() {
    static hipblasHandle_t cublas_handle;
    static bool is_intialized = false;
    if(!is_intialized) {
        auto status = hipblasCreate(&cublas_handle);

        if(status != HIPBLAS_STATUS_SUCCESS) {
            std::cerr << "error: unable to initialize cublas" << std::endl;
            exit(-1);
        }
    }

    return cublas_handle;
}

